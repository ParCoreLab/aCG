/* This file is part of acg.
 *
 * Copyright 2025 Koç University and Simula Research Laboratory
 *
 * Permission is hereby granted, free of charge, to any person
 * obtaining a copy of this software and associated documentation
 * files (the “Software”), to deal in the Software without
 * restriction, including without limitation the rights to use, copy,
 * modify, merge, publish, distribute, sublicense, and/or sell copies
 * of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS
 * BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN
 * ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 * Authors: James D. Trotter <james@simula.no>
 *
 * Last modified: 2025-04-26
 *
 * wrapper functions to provide a C API for NVSHMEM
 */

#include "acg/config.h"
#include "acg/nvshmem.h"

#if defined(ACG_HAVE_CUDA)
#include <hip/hip_runtime_api.h>
#endif

#if defined(ACG_HAVE_NVSHMEM)
#include <nvshmem.h>
#include <nvshmemx.h>
#endif

#ifdef __cplusplus
extern "C" {
#endif

#if defined(ACG_HAVE_CUDA) && defined(ACG_HAVE_NVSHMEM)

/*
 * library handles
 */

nvshmem_team_t acg_nvshmem_team(acg_nvshmem_team_t team)
{
    if (team == ACG_NVSHMEM_TEAM_INVALID) return NVSHMEM_TEAM_INVALID;
    else if (team == ACG_NVSHMEM_TEAM_WORLD) return NVSHMEM_TEAM_WORLD;
    else if (team == ACG_NVSHMEM_TEAM_SHARED) return NVSHMEM_TEAM_SHARED;
    else if (team == ACG_NVSHMEMX_TEAM_NODE) return NVSHMEMX_TEAM_NODE;
    else return team;
}

/*
 * library setup, exit, and query
 */

void acg_nvshmem_init(void) { nvshmem_init(); }
int acg_nvshmemx_init_attr(unsigned int flags, nvshmemx_init_attr_t *attributes) { return nvshmemx_init_attr(flags, attributes); }
int acg_nvshmem_my_pe(void) { return nvshmem_my_pe(); }
int acg_nvshmem_n_pes(void) { return nvshmem_n_pes(); }
void acg_nvshmem_finalize(void) { nvshmem_finalize(); }
void acg_nvshmem_info_get_version(int *major, int *minor) { nvshmem_info_get_version(major, minor); }
void acg_nvshmem_info_get_name(char *name) { nvshmem_info_get_name(name); }
#if NVSHMEM_MAJOR_VERSION >= 3
void acg_nvshmemx_vendor_get_version_info(int *major, int *minor, int *patch) { nvshmemx_vendor_get_version_info(major, minor, patch); }
#else
void acg_nvshmemx_vendor_get_version_info(int *major, int *minor, int *patch) { *major = NVSHMEM_VENDOR_MAJOR_VERSION; *minor = NVSHMEM_VENDOR_MINOR_VERSION; *patch = NVSHMEM_VENDOR_PATCH_VERSION; }
#endif

/*
 * memory management
 */

void *acg_nvshmem_malloc(size_t size) { return nvshmem_malloc(size); }
void acg_nvshmem_free(void *ptr) { nvshmem_free(ptr); }
void *acg_nvshmem_align(size_t alignment, size_t size) { return nvshmem_align(alignment, size); }
void *acg_nvshmem_calloc(size_t count, size_t size) { return nvshmem_calloc(count, size); }

/*
 * implicit team collectives
 */

void acg_nvshmem_barrier_all(void) { nvshmem_barrier_all(); }
void acg_nvshmemx_barrier_all_on_stream(hipStream_t stream) { nvshmemx_barrier_all_on_stream(stream); }
void acg_nvshmem_sync_all(void) { nvshmem_sync_all(); }
void acg_nvshmemx_sync_all_on_stream(hipStream_t stream) { nvshmemx_sync_all_on_stream(stream); }
int acg_nvshmem_double_sum_reduce(acg_nvshmem_team_t team, double *dest, const double *source, size_t nreduce) { return nvshmem_double_sum_reduce(acg_nvshmem_team(team), dest, source, nreduce); }
int acg_nvshmemx_double_sum_reduce_on_stream(acg_nvshmem_team_t team, double *dest, const double *source, size_t nreduce, hipStream_t stream) { return nvshmemx_double_sum_reduce_on_stream(acg_nvshmem_team(team), dest, source, nreduce, stream); }
#endif

#ifdef __cplusplus
}
#endif
